#include "hip/hip_runtime.h"
/*
 Copyright (c) LEADERG Co., Ltd. All rights reserved.
 Free of charge for noncommercial purposes.
 Please contact LEADERG to buy licenses for commercial purposes. (http://www.leaderg.com/license)
 THIS SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY.
 */


#include "gt_lib.h"
#include "gt_image.h"

#ifdef __cplusplus
extern "C" 
{
#endif

static int gtImageCudaIsInit = 0;

int GtImageCuda_InitLib(gt_utf8 *license, gt_utf8 *token) {
	if (gtImageCudaIsInit == 1) return GT_OK_INIT_ALREADY;
	gtImageCudaIsInit = 1;

	// Init CUDA device.
	unsigned char *deviceMemA;
	hipMalloc((void**)&deviceMemA, 1024 * sizeof(unsigned char));
	hipFree(deviceMemA);
	
	return GT_OK;
}

int GtImageCuda_Test() {
	if (gtImageCudaIsInit == 0) return GT_ERROR_NOT_INIT;

	return GT_OK;
}

__global__ void gtImageCuda_SetThreshold(unsigned char *deviceMemA, unsigned char *deviceMemB, int width, int height, int threshold) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	
	int shift = x + y * width;

	if ((x < width) && (y < height)) {
		if (deviceMemA[shift] >= threshold) {
			deviceMemB[shift] = 255;
		} else {
			deviceMemB[shift] = 0;
		}
	}
}

int GtImageCuda_SetThreshold(GtImage_t *image, int threshold) {
	if (image == NULL) return GT_ERROR_PARAMETER_0;
	if (image->magic != GT_MAGIC_IMAGE) return GT_ERROR_PARAMETER_0;
	if (threshold < 0) return GT_ERROR_PARAMETER_1;
	if (threshold > 255) return GT_ERROR_PARAMETER_1;

	if (image->colorMode != GT_COLOR_MODE_GRAY8) {
		return GT_ERROR_COLOR_MODE;
	}

	unsigned char *deviceMemA;
	unsigned char *deviceMemB;

	hipMalloc((void**)&deviceMemA, image->dataSize * sizeof(unsigned char));
	hipMemcpy(deviceMemA, image->data, image->dataSize * sizeof(unsigned char), hipMemcpyHostToDevice);

	hipMalloc((void**)&deviceMemB, image->dataSize * sizeof(unsigned char));
	
	// 1 block = 32 x 32 = 1024 threads.
	dim3 block(32, 32);

	// 1 grid = X x Y blocks. ceil is the smallest integral value that is not less than x.
	dim3 grid((size_t)(ceil( ((float)image->width)/ ((float)block.x) )),(size_t)(ceil( ((float)image->height)/ ((float)block.y) ))); 
	
	gtImageCuda_SetThreshold<<<grid,block>>>(deviceMemA, deviceMemB, image->width, image->height, threshold);

	hipMemcpy(image->data , deviceMemB, image->dataSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(deviceMemA);
	hipFree(deviceMemB);

	return GT_OK;
}

__global__ void gtImageCuda_SetBrightness(unsigned char *deviceMemA, unsigned char *deviceMemB, int width, int height, int brightness) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int shift = x + y * width;

	int temp;
	if ((x < width) && (y < height)) {
        temp = deviceMemA[shift] + brightness;
		if (temp > 255) {
			deviceMemB[shift] = 255;
		} else if (temp < 0) {
			deviceMemB[shift] = 0;
		} else {
			deviceMemB[shift] = temp;
		}
	}
}

int GtImageCuda_SetBrightness(GtImage_t *image, int brightness) {
	if (image == NULL) return GT_ERROR_PARAMETER_0;
	if (image->magic != GT_MAGIC_IMAGE) return GT_ERROR_PARAMETER_0;

	if (image->colorMode != GT_COLOR_MODE_GRAY8) {
		return GT_ERROR_COLOR_MODE;
	}

	unsigned char *deviceMemA;
	unsigned char *deviceMemB;

	hipMalloc((void**)&deviceMemA, image->dataSize * sizeof(unsigned char));
	hipMemcpy(deviceMemA, image->data, image->dataSize * sizeof(unsigned char), hipMemcpyHostToDevice);

	hipMalloc((void**)&deviceMemB, image->dataSize * sizeof(unsigned char));
	
	// 1 block = 32 x 32 = 1024 threads.
	dim3 block(32, 32);

	// 1 grid = X x Y blocks. ceil is the smallest integral value that is not less than x.
	dim3 grid((size_t)(ceil( ((float)image->width)/ ((float)block.x) )),(size_t)(ceil( ((float)image->height)/ ((float)block.y) ))); 
	
	gtImageCuda_SetBrightness<<<grid,block>>>(deviceMemA, deviceMemB, image->width, image->height, brightness);

	hipMemcpy(image->data , deviceMemB, image->dataSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(deviceMemA);
	hipFree(deviceMemB);

	return GT_OK;
}

int GtImageCuda_GetCorrelation(GtImage_t *imageRoi, GtImage_t *imagePattern, GtImage_t *imageCorrelation) {
	if (imageRoi == NULL) return GT_ERROR_PARAMETER_0;
	if (imageRoi->magic != GT_MAGIC_IMAGE) return GT_ERROR_PARAMETER_0;
	if (imagePattern == NULL) return GT_ERROR_PARAMETER_1;
	if (imagePattern->magic != GT_MAGIC_IMAGE) return GT_ERROR_PARAMETER_1;
	if (imageCorrelation == NULL) return GT_ERROR_PARAMETER_2;
	if (imageCorrelation->magic != GT_MAGIC_IMAGE) return GT_ERROR_PARAMETER_2;
	
	if (imageRoi->colorMode != GT_COLOR_MODE_GRAY8) {
		return GT_ERROR_COLOR_MODE;
	}

	if (imagePattern->colorMode != GT_COLOR_MODE_GRAY8) {
		return GT_ERROR_COLOR_MODE;
	}

	if (imageRoi->width < imagePattern->width) {
		return GT_ERROR_PARAMETER_1;
	}

	if (imageRoi->height < imagePattern->height) {
		return GT_ERROR_PARAMETER_1;
	}

	// TODO

	return GT_OK;
}

__global__ void gtImageCuda_SetContrast(unsigned char *deviceMemA, unsigned char *deviceMemB, int width, int height, float contrast) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int shift = x + y * width;

	float temp;
	if((x < width) && (y < height)) {
       temp = deviceMemA[shift] * contrast;
		if (temp > 255) {
			deviceMemB[shift] = 255;
		} else if (temp < 0) {
			deviceMemB[shift] = 0;
		} else {
			deviceMemB[shift] = (unsigned char)temp;
		}
	}
}


int GtImageCuda_SetContrast(GtImage_t *image, float contrast) {
	if (image == NULL) return GT_ERROR_PARAMETER_0;
	if (image->magic != GT_MAGIC_IMAGE) return GT_ERROR_PARAMETER_0;

	if (image->colorMode != GT_COLOR_MODE_GRAY8) {
		return GT_ERROR_COLOR_MODE;
	}

	unsigned char *deviceMemA;
	unsigned char *deviceMemB;

	hipMalloc((void**)&deviceMemA, image->dataSize * sizeof(unsigned char));
	hipMemcpy(deviceMemA, image->data, image->dataSize * sizeof(unsigned char), hipMemcpyHostToDevice);

	hipMalloc((void**)&deviceMemB, image->dataSize * sizeof(unsigned char));

	// 1 block = 32 x 32 = 1024 threads.
	dim3 block(32, 32);

	// 1 grid = X x Y blocks. ceil is the smallest integral value that is not less than x.
	dim3 grid((size_t)(ceil( ((float)image->width)/ ((float)block.x) )),(size_t)(ceil( ((float)image->height)/ ((float)block.y) )));

	gtImageCuda_SetContrast<<<grid,block>>>(deviceMemA, deviceMemB, image->width, image->height, contrast);

	hipMemcpy(image->data , deviceMemB, image->dataSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(deviceMemA);
	hipFree(deviceMemB);

	return GT_OK;
}


__global__ void gtImageCuda_Invert(unsigned char *deviceMemA, unsigned char *deviceMemB, int width, int height) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int shift = x + y * width;

	if((x < width) && (y < height))
		deviceMemB[shift] = 255 - abs(deviceMemA[shift] % 255);
}


int GtImageCuda_Invert(GtImage_t *image) {
	if (image == NULL) return GT_ERROR_PARAMETER_0;
	if (image->magic != GT_MAGIC_IMAGE) return GT_ERROR_PARAMETER_0;

	if (image->colorMode != GT_COLOR_MODE_GRAY8) {
		return GT_ERROR_COLOR_MODE;
	}

	unsigned char *deviceMemA;
	unsigned char *deviceMemB;

	hipMalloc((void**)&deviceMemA, image->dataSize * sizeof(unsigned char));
	hipMemcpy(deviceMemA, image->data, image->dataSize * sizeof(unsigned char), hipMemcpyHostToDevice);

	hipMalloc((void**)&deviceMemB, image->dataSize * sizeof(unsigned char));

	// 1 block = 32 x 32 = 1024 threads.
	dim3 block(32, 32);

	// 1 grid = X x Y blocks. ceil is the smallest integral value that is not less than x.
	dim3 grid((size_t)(ceil( ((float)image->width)/ ((float)block.x) )),(size_t)(ceil( ((float)image->height)/ ((float)block.y) )));

	gtImageCuda_Invert<<<grid,block>>>(deviceMemA, deviceMemB, image->width, image->height);

	hipMemcpy(image->data , deviceMemB, image->dataSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(deviceMemA);
	hipFree(deviceMemB);

	return GT_OK;
}

#ifdef __cplusplus
}
#endif
